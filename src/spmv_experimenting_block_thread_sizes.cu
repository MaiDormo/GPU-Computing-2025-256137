#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>

#include "../include/read_file_lib.h"
#include "../include/spmv_type.h"
#include "../include/csr_conversion.h"
#include "../include/spmv_utils.h"
#include "../include/spmv_kernels.h"

// --- Main Function ---
int main(int argc, char ** argv) { 
    
    if (argc < 2 || argc > 5) {
    printf("Usage: <./bin/spmv_experimenting_block_thread_sizes> <path/to/file.mtx> [BLOCK_SIZE] [BLOCK_NUM]\n");
    return -1;
    }

    // default
    int block_size = 256;
    int block_num = -1;

    if (argc >= 3) block_size = atoi(argv[2]);
    if (argc == 4) block_num = atoi(argv[3]);

    // --- Host Data Structures ---
    struct COO h_coo;
    struct CSR h_csr;
    dtype *h_vec = NULL;
    dtype *h_res = NULL;

    // --- Read Matrix ---
    read_from_file_and_init(argv[1], &h_coo);
    int n = h_coo.num_rows;
    int m = h_coo.num_cols;
    int nnz = h_coo.num_non_zeros;

    // --- Allocate Host Memory ---
    h_vec = (dtype*)malloc(m * sizeof(dtype));
    h_res = (dtype*)malloc(n * sizeof(dtype));
    h_csr.values = (dtype*)malloc(nnz * sizeof(dtype));
    h_csr.col_indices = (int*)malloc(nnz * sizeof(int));
    h_csr.row_pointers = (int*)calloc(n + 1, sizeof(int)); // Zero initialization is important

    if (!h_vec || !h_res || !h_csr.values || !h_csr.col_indices || !h_csr.row_pointers) {
        perror("Failed to allocate host memory");
        // Free any successfully allocated memory before exiting
        free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
        free(h_vec); free(h_res);
        free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);
        return -1;
    }

    // --- Initialize Host Vectors ---
    for (int i = 0; i < m; i++) h_vec[i] = 1.0;
    memset(h_res, 0, n * sizeof(dtype));

    // --- Convert COO to CSR ---
    if (coo_to_csr(&h_coo, &h_csr) != 0) {
         fprintf(stderr, "Error during COO to CSR conversion.\n");
         // Free memory and exit
         free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
         free(h_vec); free(h_res);
         free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);;
         return -1;
    }

    // Free original COO data (host)
    free(h_coo.a_val); h_coo.a_val = NULL;
    free(h_coo.a_row); h_coo.a_row = NULL;
    free(h_coo.a_col); h_coo.a_col = NULL;

    // --- Device Data Structures ---
    struct CSR d_csr; // Holds device pointers
    dtype *d_vec = NULL, *d_res = NULL;

    // --- Allocate Device Memory ---
    hipMalloc(&d_vec, m * sizeof(dtype));
    hipMalloc(&d_res, n * sizeof(dtype));
    hipMalloc(&d_csr.values, nnz * sizeof(dtype));
    hipMalloc(&d_csr.col_indices, nnz * sizeof(int));
    hipMalloc(&d_csr.row_pointers, (n + 1) * sizeof(int));
    d_csr.num_rows = n;
    d_csr.num_cols = m;
    d_csr.num_non_zeros = nnz;

    // --- Copy Data to Device ---
    hipMemcpy(d_vec, h_vec, m * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_res, h_res, n * sizeof(dtype), hipMemcpyHostToDevice); // Copy initial zeros
    hipMemcpy(d_csr.values, h_csr.values, nnz * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.col_indices, h_csr.col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.row_pointers, h_csr.row_pointers, (n + 1) * sizeof(int), hipMemcpyHostToDevice);

    // --- Kernel Launch Configuration ---
    if (block_num <= 0) {
        // Default: enough blocks to cover all nnz with 1 element per thread
        block_num = (nnz + block_size - 1) / block_size;
    }
    int total_threads = block_size * block_num;
    int elements_per_thread = (nnz + total_threads - 1) / total_threads;

    

    // --- Timing Setup ---
    const int NUM_RUNS = 50;
    dtype total_time = 0.0;
    dtype times[NUM_RUNS];
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // --- Warmup Run ---
    value_parallel_blocked_spmv<<<block_num, block_size>>>(
        d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, d_csr.num_non_zeros, n, elements_per_thread
    );
    
    hipDeviceSynchronize();
    
    // --- Timed Runs ---
    for (int run = 0; run < NUM_RUNS; run++) {
        // Reset result vector before each run
        hipMemset(d_res, 0, n * sizeof(dtype));

        hipEventRecord(start);

        value_parallel_blocked_spmv<<<block_num, block_size>>>(
            d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, d_csr.num_non_zeros, n, elements_per_thread
        );
            
        hipEventRecord(end);
        hipEventSynchronize(end);

        float millisec = 0.0;
        hipEventElapsedTime(&millisec, start, end);
        times[run] = millisec * 1e-3;
    }

    // --- Copy Result Back ---
    hipMemcpy(h_res, d_res, n * sizeof(dtype), hipMemcpyDeviceToHost);

    // --- Performance Calculation ---
    hipEventDestroy(start);
    hipEventDestroy(end);

    for (int i = 0; i < NUM_RUNS; i++) {
        total_time += times[i];
    }
    dtype avg_time = total_time / NUM_RUNS;

    double bandwidth, gflops;
    calculate_bandwidth(n,m,nnz,h_csr.col_indices, avg_time, &bandwidth, &gflops);
    
    // --- Print Results ---
    print_spmv_performance(
        "CSR", 
        argv[1],
        n, 
        m, 
        nnz, 
        avg_time, 
        bandwidth, 
        gflops, 
        h_res,
        10  // Print up to 10 samples
    );

    // --- Cleanup ---
    hipFree(d_vec);
    hipFree(d_res);
    hipFree(d_csr.values);
    hipFree(d_csr.col_indices);
    hipFree(d_csr.row_pointers);

    free(h_vec);
    free(h_res);
    free(h_csr.values);
    free(h_csr.col_indices);
    free(h_csr.row_pointers);


    return 0;
}