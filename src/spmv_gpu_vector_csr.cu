#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>

#include "../include/read_file_lib.h"
#include "../include/spmv_type.h"
#include "../include/csr_conversion.h"
#include "../include/spmv_utils.h"
#include "../include/spmv_kernels.h"

#define WARP_SIZE 32

// --- Main Function ---
int main(int argc, char ** argv) { 
    
    if (argc != 2) {
        printf("Usage: <./bin/spmv_*> <path/to/file.mtx>\n");
        return -1;
    }

    // --- Host Data Structures ---
    struct COO h_coo;
    struct CSR h_csr;
    dtype *h_vec = NULL;
    dtype *h_res = NULL;

    // --- Read Matrix ---
    read_from_file_and_init(argv[1], &h_coo);
    int n = h_coo.num_rows;
    int m = h_coo.num_cols;
    int nnz = h_coo.num_non_zeros;

    // --- Allocate Host Memory ---
    h_vec = (dtype*)malloc(m * sizeof(dtype));
    h_res = (dtype*)malloc(n * sizeof(dtype));
    h_csr.values = (dtype*)malloc(nnz * sizeof(dtype));
    h_csr.col_indices = (int*)malloc(nnz * sizeof(int));
    h_csr.row_pointers = (int*)calloc(n + 1, sizeof(int)); // Zero initialization is important

    if (!h_vec || !h_res || !h_csr.values || !h_csr.col_indices || !h_csr.row_pointers) {
        perror("Failed to allocate host memory");
        // Free any successfully allocated memory before exiting
        free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
        free(h_vec); free(h_res);
        free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);
        return -1;
    }

    // --- Initialize Host Vectors ---
    for (int i = 0; i < m; i++) h_vec[i] = 1.0;
    memset(h_res, 0, n * sizeof(dtype));

    // --- Convert COO to CSR ---
    if (coo_to_csr(&h_coo, &h_csr) != 0) {
         fprintf(stderr, "Error during COO to CSR conversion.\n");
         // Free memory and exit
         free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
         free(h_vec); free(h_res);
         free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);
         return -1;
    }

    // Free original COO data (host)
    free(h_coo.a_val); h_coo.a_val = NULL;
    free(h_coo.a_row); h_coo.a_row = NULL;
    free(h_coo.a_col); h_coo.a_col = NULL;

    // --- Calculate Optimal Launch Configuration ---
    int optimal_block_size;
    
    // Analyze matrix characteristics to determine optimal block size
    struct MAT_STATS stats = calculate_matrix_stats(&h_csr);
    printf("Matrix analysis: mean_nnz_per_row = %.2f\n", stats.mean_nnz_per_row);
    
    // Determine block size based on matrix density
    if (stats.mean_nnz_per_row < 16) {
        optimal_block_size = 128;  // Very sparse matrices - smaller blocks
    } else if (stats.mean_nnz_per_row < 32) {
        optimal_block_size = 256;  // Sparse matrices
    } else if (stats.mean_nnz_per_row < 64) {
        optimal_block_size = 512;  // Medium density matrices
    } else {
        optimal_block_size = 1024; // Dense matrices - larger blocks for better occupancy
    }

    // Check device limits and adjust if necessary
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    
    if (optimal_block_size > prop.maxThreadsPerBlock) {
        printf("Warning: Optimal block size (%d) exceeds device limit (%d)\n", 
               optimal_block_size, prop.maxThreadsPerBlock);
        optimal_block_size = prop.maxThreadsPerBlock;
    }

    // Ensure block size is a multiple of warp size
    optimal_block_size = ((optimal_block_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;

    // Calculate launch configuration for vector CSR double buffer kernel
    int warps_per_block = optimal_block_size / WARP_SIZE;
    int rows_per_block = warps_per_block * 2;  // 2 rows per warp in double buffer kernel
    int optimal_block_num = (n + rows_per_block - 1) / rows_per_block;


    // Ensure minimum occupancy
    // int min_blocks = prop.multiProcessorCount;
    // if (optimal_block_num < min_blocks) {
    //     optimal_block_num = min_blocks;
    // }

    // // Cap maximum blocks to avoid overhead
    // int max_blocks = prop.multiProcessorCount * 8;
    // if (optimal_block_num > max_blocks) {
    //     optimal_block_num = max_blocks;
    // }

    printf("Device properties: %d SMs, %d max threads per block\n", 
           prop.multiProcessorCount, prop.maxThreadsPerBlock);
    printf("Calculated launch config:\n");
    printf("  Block size: %d threads (%d warps)\n", optimal_block_size, warps_per_block);
    printf("  Grid size: %d blocks\n", optimal_block_num);
    printf("  Rows per block: %d (2 rows per warp)\n", rows_per_block);

    // --- Device Data Structures ---
    struct CSR d_csr; // Holds device pointers
    dtype *d_vec = NULL, *d_res = NULL;

    // --- Allocate Device Memory ---
    hipMalloc(&d_vec, m * sizeof(dtype));
    hipMalloc(&d_res, n * sizeof(dtype));
    hipMalloc(&d_csr.values, h_csr.num_non_zeros * sizeof(dtype));
    hipMalloc(&d_csr.col_indices, h_csr.num_non_zeros * sizeof(int));
    hipMalloc(&d_csr.row_pointers, (n + 1) * sizeof(int));
    
    d_csr.num_rows = n;
    d_csr.num_cols = m;
    d_csr.num_non_zeros = h_csr.num_non_zeros;

    // Check for CUDA errors
    hipError_t cuda_err = hipGetLastError();
    if (cuda_err != hipSuccess) {
        fprintf(stderr, "CUDA memory allocation failed: %s\n", hipGetErrorString(cuda_err));
        return -1;
    }

    // --- Copy Data to Device ---
    hipMemcpy(d_vec, h_vec, m * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_res, h_res, n * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.values, h_csr.values, h_csr.num_non_zeros * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.col_indices, h_csr.col_indices, h_csr.num_non_zeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.row_pointers, h_csr.row_pointers, (n + 1) * sizeof(int), hipMemcpyHostToDevice);

    // --- Timing Setup ---
    const int NUM_RUNS = 50;
    dtype total_time = 0.0;
    dtype times[NUM_RUNS];
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // --- Warmup Run ---
    vector_csr_double_buffer<<<optimal_block_num, optimal_block_size>>>(
        d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n
    );
    
    hipDeviceSynchronize();
    
    // Check for kernel launch errors
    cuda_err = hipGetLastError();
    if (cuda_err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cuda_err));
        return -1;
    }
    
    // --- Timed Runs ---
    for (int run = 0; run < NUM_RUNS; run++) {
        // Reset result vector to ensure correctness
        // hipMemset(d_res, 0, n * sizeof(dtype));
        
        // Add synchronization before starting timing
        hipDeviceSynchronize();

        hipEventRecord(start);

        vector_csr_double_buffer<<<optimal_block_num, optimal_block_size>>>(
            d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n
        );
            
        hipEventRecord(end);
        hipEventSynchronize(end);

        // Check for kernel execution errors
        cuda_err = hipGetLastError();
        if (cuda_err != hipSuccess) {
            fprintf(stderr, "Kernel execution failed at run %d: %s\n", run, hipGetErrorString(cuda_err));
            break;
        }

        float millisec = 0.0;
        hipEventElapsedTime(&millisec, start, end);
        times[run] = millisec * 1e-3;
    }

    // --- Copy Result Back ---
    hipMemcpy(h_res, d_res, n * sizeof(dtype), hipMemcpyDeviceToHost);

    // --- Performance Calculation ---
    hipEventDestroy(start);
    hipEventDestroy(end);

    for (int i = 0; i < NUM_RUNS; i++) {
        total_time += times[i];
    }
    dtype avg_time = total_time / NUM_RUNS;

    // Calculate memory bandwidth more accurately for SpMV
    // For CSR SpMV, memory access pattern is:
    // 1. Read all row_pointers (accessed sequentially)
    // 2. Read all values and col_indices (accessed sequentially) 
    // 3. Read vector elements (potentially random access pattern)
    // 4. Write result vector (sequential)

    size_t bytes_read_vals = (size_t)nnz * sizeof(dtype);           // matrix values
    size_t bytes_read_cols = (size_t)nnz * sizeof(int);            // column indices  
    size_t bytes_read_row_ptr = (size_t)(n + 1) * sizeof(int);     // row pointers
    
    // For vector reads, each column index causes a vector element read
    // This gives a more realistic bandwidth estimate for custom kernels
    // that may not have sophisticated caching mechanisms
    size_t bytes_read_vec = (size_t)nnz * sizeof(dtype);           // vector reads (one per nnz)
    
    size_t bytes_written = (size_t)n * sizeof(dtype);              // result vector
    
    // Total memory traffic
    size_t total_bytes = bytes_read_vals + bytes_read_cols + 
                        bytes_read_row_ptr + bytes_read_vec + bytes_written;

    // Memory bandwidth calculation
    double bandwidth = total_bytes / (avg_time * 1.0e9);  // GB/s
    
    // Computational intensity
    double flops = 2.0 * nnz;  // Each non-zero: 1 multiply + 1 add
    double gflops = flops / (avg_time * 1.0e9);  // GFLOPS
    
    // Calculate arithmetic intensity for roofline analysis
    double arithmetic_intensity = flops / (double)total_bytes;  // FLOPS/Byte

    // --- Print Matrix Statistics ---
    print_matrix_stats(&h_csr);

    // --- Print Results with Additional Metrics ---
    printf("\n=== Vector CSR Performance Results ===\n");
    printf("Matrix: %s\n", argv[1]);
    printf("Dimensions: %d x %d, NNZ: %d\n", n, m, nnz);
    printf("Average time: %.6f seconds\n", avg_time);
    printf("Memory bandwidth: %.2f GB/s\n", bandwidth);
    printf("Compute performance: %.2f GFLOPS\n", gflops);
    printf("Arithmetic intensity: %.3f FLOPS/Byte\n", arithmetic_intensity);
    printf("Memory breakdown:\n");
    printf("  Matrix values: %.2f MB\n", bytes_read_vals / (1024.0 * 1024.0));
    printf("  Column indices: %.2f MB\n", bytes_read_cols / (1024.0 * 1024.0));
    printf("  Row pointers: %.2f MB\n", bytes_read_row_ptr / (1024.0 * 1024.0));
    printf("  Vector reads: %.2f MB\n", bytes_read_vec / (1024.0 * 1024.0));
    printf("  Result writes: %.2f MB\n", bytes_written / (1024.0 * 1024.0));
    printf("  Total memory: %.2f MB\n", total_bytes / (1024.0 * 1024.0));

    // Also call the standard print function for consistency
    print_spmv_performance(
        "Vector CSR Double Buffer", 
        argv[1],
        n, 
        m, 
        nnz, 
        avg_time, 
        bandwidth, 
        gflops, 
        h_res,
        10  // Print up to 10 samples
    );

    // --- Cleanup ---
    hipFree(d_vec);
    hipFree(d_res);
    hipFree(d_csr.values);
    hipFree(d_csr.col_indices);
    hipFree(d_csr.row_pointers);

    free(h_vec);
    free(h_res);
    free(h_csr.values);
    free(h_csr.col_indices);
    free(h_csr.row_pointers);

    return 0;
}