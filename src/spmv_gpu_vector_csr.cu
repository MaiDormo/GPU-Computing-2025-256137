#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>

#include "../include/read_file_lib.h"
#include "../include/spmv_type.h"
#include "../include/csr_conversion.h"
#include "../include/spmv_utils.h"
#include "../include/spmv_kernels.h"

// --- Main Function ---
int main(int argc, char ** argv) { 
    
    if (argc != 2) {
        printf("Usage: <./bin/spmv_*> <path/to/file.mtx>\n");
        return -1;
    }

    // --- Host Data Structures ---
    struct COO h_coo;
    struct CSR h_csr;
    dtype *h_vec = NULL;
    dtype *h_res = NULL;

    // --- Read Matrix ---
    read_from_file_and_init(argv[1], &h_coo);
    int n = h_coo.num_rows;
    int m = h_coo.num_cols;
    int nnz = h_coo.num_non_zeros;

    // --- Allocate Host Memory ---
    h_vec = (dtype*)malloc(m * sizeof(dtype));
    h_res = (dtype*)malloc(n * sizeof(dtype));
    h_csr.values = (dtype*)malloc(nnz * sizeof(dtype));
    h_csr.col_indices = (int*)malloc(nnz * sizeof(int));
    h_csr.row_pointers = (int*)calloc(n + 1, sizeof(int)); // Zero initialization is important

    if (!h_vec || !h_res || !h_csr.values || !h_csr.col_indices || !h_csr.row_pointers) {
        perror("Failed to allocate host memory");
        // Free any successfully allocated memory before exiting
        free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
        free(h_vec); free(h_res);
        free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);
        return -1;
    }

    // --- Initialize Host Vectors ---
    for (int i = 0; i < m; i++) h_vec[i] = 1.0;
    memset(h_res, 0, n * sizeof(dtype));

    // --- Convert COO to CSR ---
    if (coo_to_csr(&h_coo, &h_csr) != 0) {
         fprintf(stderr, "Error during COO to CSR conversion.\n");
         // Free memory and exit
         free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
         free(h_vec); free(h_res);
         free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);
         return -1;
    }

    // Free original COO data (host)
    free(h_coo.a_val); h_coo.a_val = NULL;
    free(h_coo.a_row); h_coo.a_row = NULL;
    free(h_coo.a_col); h_coo.a_col = NULL;

    // --- Device Data Structures ---
    struct CSR d_csr; // Holds device pointers
    dtype *d_vec = NULL, *d_res = NULL;

    // --- Allocate Device Memory ---
    hipMalloc(&d_vec, m * sizeof(dtype));
    hipMalloc(&d_res, n * sizeof(dtype));
    hipMalloc(&d_csr.values, h_csr.num_non_zeros * sizeof(dtype));
    hipMalloc(&d_csr.col_indices, h_csr.num_non_zeros * sizeof(int));
    hipMalloc(&d_csr.row_pointers, (n + 1) * sizeof(int));
    d_csr.num_rows = n;
    d_csr.num_cols = m;
    d_csr.num_non_zeros = h_csr.num_non_zeros;

    // --- Copy Data to Device ---
    hipMemcpy(d_vec, h_vec, m * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_res, h_res, n * sizeof(dtype), hipMemcpyHostToDevice); // Copy initial zeros
    hipMemcpy(d_csr.values, h_csr.values, h_csr.num_non_zeros * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.col_indices, h_csr.col_indices, h_csr.num_non_zeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.row_pointers, h_csr.row_pointers, (n + 1) * sizeof(int), hipMemcpyHostToDevice);

    // --- Kernel Launch Configuration --- 
    const int warps_per_block = BLOCK_SIZE/ WARP_SIZE;
    const int rows_per_block = warps_per_block;
    const int block_num = (n + rows_per_block - 1) / rows_per_block;

    // --- Timing Setup ---
    const int NUM_RUNS = 50;
    dtype total_time = 0.0;
    dtype times[NUM_RUNS];
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // --- Warmup Run ---
    vector_csr<<<block_num, BLOCK_SIZE>>>(
        d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n
    );
    
    hipDeviceSynchronize();
    
    // --- Timed Runs ---
    for (int run = 0; run < NUM_RUNS; run++) {
        hipEventRecord(start);

        vector_csr<<<block_num, BLOCK_SIZE>>>(
            d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n
        );
            
        hipEventRecord(end);
        hipEventSynchronize(end);

        float millisec = 0.0;
        hipEventElapsedTime(&millisec, start, end);
        times[run] = millisec * 1e-3;
    }

    // --- Copy Result Back ---
    hipMemcpy(h_res, d_res, n * sizeof(dtype), hipMemcpyDeviceToHost);

    // --- Performance Calculation ---
    hipEventDestroy(start);
    hipEventDestroy(end);

    for (int i = 0; i < NUM_RUNS; i++) {
        total_time += times[i];
    }
    dtype avg_time = total_time / NUM_RUNS;

    // Calculate effective memory access more accurately
    size_t bytes_read_vals_cols = (size_t)nnz * (sizeof(dtype) + sizeof(int)); // values and col indices
    size_t bytes_read_row_ptr = (size_t)(n + 1) * sizeof(int);                // row pointers

    // Count unique column indices
    int* unique_cols = (int*)calloc(m, sizeof(int));
    size_t unique_count = 0;
    for (int i = 0; i < nnz; i++) {
        if (unique_cols[h_csr.col_indices[i]] == 0) {
            unique_cols[h_csr.col_indices[i]] = 1;
            unique_count++;
        }
    }
    free(unique_cols);

    // Use the actual count of unique elements
    size_t bytes_read_vec = unique_count * sizeof(dtype);

    size_t bytes_read = bytes_read_vals_cols + // values and col indices
                        bytes_read_row_ptr +               // row pointers
                        bytes_read_vec;                   // vector reads (worst case estimate)
    size_t bytes_written = (size_t)n * sizeof(dtype);                 // result vector
    size_t total_bytes = bytes_read + bytes_written;

    double bandwidth = total_bytes / (avg_time * 1.0e9);  // GB/s
    double flops = 2.0 * nnz;
    double gflops = flops / (avg_time * 1.0e9);  // GFLOPS

    // --- Print Results ---
    print_spmv_performance(
        "CSR", 
        argv[1],
        n, 
        m, 
        nnz, 
        avg_time, 
        bandwidth, 
        gflops, 
        h_res,
        10  // Print up to 10 samples
    );

    // --- Cleanup ---
    hipFree(d_vec);
    hipFree(d_res);
    hipFree(d_csr.values);
    hipFree(d_csr.col_indices);
    hipFree(d_csr.row_pointers);

    free(h_vec);
    free(h_res);
    free(h_csr.values);
    free(h_csr.col_indices);
    free(h_csr.row_pointers);

    return 0;
}