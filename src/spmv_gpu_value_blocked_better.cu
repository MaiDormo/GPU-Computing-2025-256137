#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>

#include "../include/read_file_lib.h"
#include "../include/spmv_type.h"
#include "../include/csr_conversion.h"
#include "../include/spmv_utils.h"
#include "../include/spmv_kernels.h"

// --- Main Function ---
int main(int argc, char ** argv) { 
    
    if (argc != 2) {
        printf("Usage: <./bin/spmv_*> <path/to/file.mtx>\n");
        return -1;
    }

    // --- Host Data Structures ---
    struct COO h_coo;
    struct CSR h_csr;
    dtype *h_vec = NULL;
    dtype *h_res = NULL;

    // --- Read Matrix ---
    read_from_file_and_init(argv[1], &h_coo);
    int n = h_coo.num_rows;
    int m = h_coo.num_cols;
    int nnz = h_coo.num_non_zeros;

    // --- Allocate Host Memory ---
    h_vec = (dtype*)malloc(m * sizeof(dtype));
    h_res = (dtype*)malloc(n * sizeof(dtype));
    h_csr.values = (dtype*)malloc(nnz * sizeof(dtype));
    h_csr.col_indices = (int*)malloc(nnz * sizeof(int));
    h_csr.row_pointers = (int*)calloc(n + 1, sizeof(int)); // Zero initialization is important

    if (!h_vec || !h_res || !h_csr.values || !h_csr.col_indices || !h_csr.row_pointers) {
        perror("Failed to allocate host memory");
        // Free any successfully allocated memory before exiting
        free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
        free(h_vec); free(h_res);
        free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);
        return -1;
    }

    // --- Initialize Host Vectors ---
    for (int i = 0; i < m; i++) h_vec[i] = 1.0;
    memset(h_res, 0, n * sizeof(dtype));

    // --- Convert COO to CSR ---
    if (coo_to_csr(&h_coo, &h_csr) != 0) {
         fprintf(stderr, "Error during COO to CSR conversion.\n");
         // Free memory and exit
         free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
         free(h_vec); free(h_res);
         free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);
         return -1;
    }

    // Free original COO data (host)
    free(h_coo.a_val); h_coo.a_val = NULL;
    free(h_coo.a_row); h_coo.a_row = NULL;
    free(h_coo.a_col); h_coo.a_col = NULL;

    // --- Device Data Structures ---
    struct CSR d_csr; // Holds device pointers
    dtype *d_vec = NULL, *d_res = NULL;

    // --- Allocate Device Memory ---
    hipMalloc(&d_vec, m * sizeof(dtype));
    hipMalloc(&d_res, n * sizeof(dtype));
    hipMalloc(&d_csr.values, nnz * sizeof(dtype));
    hipMalloc(&d_csr.col_indices, nnz * sizeof(int));
    hipMalloc(&d_csr.row_pointers, (n + 1) * sizeof(int));
    d_csr.num_rows = n;
    d_csr.num_cols = m;
    d_csr.num_non_zeros = nnz;

    // --- Copy Data to Device ---
    hipMemcpy(d_vec, h_vec, m * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_res, h_res, n * sizeof(dtype), hipMemcpyHostToDevice); // Copy initial zeros
    hipMemcpy(d_csr.values, h_csr.values, nnz * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.col_indices, h_csr.col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.row_pointers, h_csr.row_pointers, (n + 1) * sizeof(int), hipMemcpyHostToDevice);

    // --- Adaptive Block Size Selection ---
    int adaptive_block_size;
    double avg_nnz_per_row = (double)nnz / n;
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    // For extremely sparse matrices (< 2 NNZ per row)
    if (avg_nnz_per_row < 2.0) {
        adaptive_block_size = 128;  // Smaller blocks for better occupancy with sparse data
    } else if (avg_nnz_per_row < 8.0) {
        adaptive_block_size = 256;  // Medium sparse
    } else if (avg_nnz_per_row < 32.0) {
        adaptive_block_size = 512;  // Medium dense
    } else {
        adaptive_block_size = 1024; // Dense matrices
    }

    // Ensure it's a multiple of warp size and within device limits
    adaptive_block_size = ((adaptive_block_size + 31) / 32) * 32;
    if (adaptive_block_size > prop.maxThreadsPerBlock) {
        adaptive_block_size = prop.maxThreadsPerBlock;
    }

    printf("Matrix analysis: %.2f avg NNZ per row\n", avg_nnz_per_row);
    printf("Selected adaptive block size: %d (was %d)\n", adaptive_block_size, BLOCK_SIZE);

    // --- Updated Kernel Launch Configuration ---
    const int elements_per_thread = 8;
    const int total_threads_needed = (nnz + elements_per_thread - 1) / elements_per_thread;
    const int block_num = (total_threads_needed + adaptive_block_size - 1) / adaptive_block_size;

    printf("Launch configuration: %d blocks, %d threads per block\n", block_num, adaptive_block_size);
    printf("Elements per thread: %d, Total threads needed: %d\n", elements_per_thread, total_threads_needed);

    // --- Timing Setup ---
    const int NUM_RUNS = 50;
    dtype total_time = 0.0;
    dtype times[NUM_RUNS];
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // --- Warmup Run ---
    value_parallel_blocked_spmv_v3<<<block_num, adaptive_block_size>>>(
        d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, d_csr.num_non_zeros, n, elements_per_thread
    );
    
    hipDeviceSynchronize();
    
    // --- Timed Runs ---
    for (int run = 0; run < NUM_RUNS; run++) {
        // Reset result vector before each run
        hipMemset(d_res, 0, n * sizeof(dtype));

        hipEventRecord(start);

        value_parallel_blocked_spmv_v3<<<block_num, adaptive_block_size>>>(
            d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, d_csr.num_non_zeros, n, elements_per_thread
        );
            
        hipEventRecord(end);
        hipEventSynchronize(end);

        float millisec = 0.0;
        hipEventElapsedTime(&millisec, start, end);
        times[run] = millisec * 1e-3;
    }

    // --- Copy Result Back ---
    hipMemcpy(h_res, d_res, n * sizeof(dtype), hipMemcpyDeviceToHost);

    // --- Performance Calculation ---
    hipEventDestroy(start);
    hipEventDestroy(end);

    for (int i = 0; i < NUM_RUNS; i++) {
        total_time += times[i];
    }
    dtype avg_time = total_time / NUM_RUNS;

    double bandwidth, gflops;
    calculate_bandwidth(n,m,nnz,h_csr.col_indices, avg_time, &bandwidth, &gflops);

    // --- Print Results ---
    print_spmv_performance(
        "Value Parallel Blocked v3 (Hash Table)", 
        argv[1],
        n, 
        m, 
        nnz, 
        avg_time, 
        bandwidth, 
        gflops, 
        h_res,
        10  // Print up to 10 samples
    );

    // --- Cleanup ---
    hipFree(d_vec);
    hipFree(d_res);
    hipFree(d_csr.values);
    hipFree(d_csr.col_indices);
    hipFree(d_csr.row_pointers);

    free(h_vec);
    free(h_res);
    free(h_csr.values);
    free(h_csr.col_indices);
    free(h_csr.row_pointers);

    return 0;
}