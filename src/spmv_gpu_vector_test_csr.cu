#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>

#include "../include/read_file_lib.h"
#include "../include/spmv_type.h"
#include "../include/csr_conversion.h"
#include "../include/spmv_utils.h"
#include "../include/spmv_kernels.h"

#define WARP_SIZE 32

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))

void calculate_advanced_launch_config(const struct CSR *csr,
                                      int *block_size,
                                      int *grid_size,
                                      int *shared_mem_size,
                                      int *kernel_variant) {
    struct MAT_STATS stats = calculate_matrix_stats(csr);
    double avg_nnz = stats.mean_nnz_per_row;
    double std_nnz = stats.std_dev_nnz_per_row;
    double cv      = (avg_nnz > 0.0) ? std_nnz / avg_nnz : 0.0;

    // Get device properties first
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    // ----- Select kernel variant & block size -----
    if (avg_nnz < 4.0) {
        *kernel_variant  = 0;
        *block_size      = 128;
    } else if (avg_nnz < 16.0) {
        *kernel_variant  = 2;
        *block_size      = 256;
    } else if (avg_nnz < 64.0) {
        if (cv > 1.0) {
            *kernel_variant  = 1;
            *block_size      = 512;
        } else {
            *kernel_variant  = 2;
            *block_size      = 512;
        }
    } else {
        *kernel_variant  = 2;
        *block_size      = 1024;
    }

    // Enforce device limits on block size
    *block_size = MIN(*block_size, prop.maxThreadsPerBlock);
    *block_size = ((*block_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;

    // ----- Compute grid size correctly -----
    int warps_per_block = *block_size / WARP_SIZE;
    int rows_per_block  = (*kernel_variant == 2) ? warps_per_block * 2
                                                 : warps_per_block;
    
    // Calculate REQUIRED grid size to cover all rows
    int required_grid_size = (csr->num_rows + rows_per_block - 1) / rows_per_block;
    
    // Set reasonable limits based on device capabilities
    int min_blocks = prop.multiProcessorCount;
    
    // Use the required grid size, but enforce minimum for occupancy
    *grid_size = MAX(required_grid_size, min_blocks);

    *shared_mem_size = (*kernel_variant == 1) ? (512 * sizeof(dtype)) : 0;
    
    // Check shared memory limits
    if (*shared_mem_size > (int)prop.sharedMemPerBlock) {
        *shared_mem_size = 0;
        if (*kernel_variant == 1) *kernel_variant = 0;
    }

    // Verify coverage
    int total_rows_covered = *grid_size * rows_per_block;
    printf("Calculated launch config: Grid=%d, Block=%d, SharedMem=%d, Variant=%d\n",
           *grid_size, *block_size, *shared_mem_size, *kernel_variant);
    printf("Coverage: %d blocks × %d rows/block = %d total rows (need %d)\n",
           *grid_size, rows_per_block, total_rows_covered, csr->num_rows);
    
    if (total_rows_covered < csr->num_rows) {
        printf("ERROR: Insufficient grid size! Only covering %d/%d rows (%.1f%%)\n",
               total_rows_covered, csr->num_rows, 
               100.0 * total_rows_covered / csr->num_rows);
    }
}

int main(int argc, char ** argv) { 
    
    if (argc != 2) {
        printf("Usage: <./bin/spmv_*> <path/to/file.mtx>\n");
        return -1;
    }

    // --- Host Data Structures ---
    struct COO h_coo;
    struct CSR h_csr;
    dtype *h_vec = NULL;
    dtype *h_res = NULL;

    // --- Read Matrix ---
    read_from_file_and_init(argv[1], &h_coo);
    int n = h_coo.num_rows;
    int m = h_coo.num_cols;
    int nnz = h_coo.num_non_zeros;

    // --- Allocate Host Memory ---
    h_vec = (dtype*)malloc(m * sizeof(dtype));
    h_res = (dtype*)malloc(n * sizeof(dtype));
    h_csr.values = (dtype*)malloc(nnz * sizeof(dtype));
    h_csr.col_indices = (int*)malloc(nnz * sizeof(int));
    h_csr.row_pointers = (int*)calloc(n + 1, sizeof(int)); // Zero initialization is important

    if (!h_vec || !h_res || !h_csr.values || !h_csr.col_indices || !h_csr.row_pointers) {
        perror("Failed to allocate host memory");
        // Free any successfully allocated memory before exiting
        free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
        free(h_vec); free(h_res);
        free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);
        return -1;
    }

    // --- Initialize Host Vectors ---
    for (int i = 0; i < m; i++) h_vec[i] = 1.0;
    memset(h_res, 0, n * sizeof(dtype));

    // --- Convert COO to CSR ---
    if (coo_to_csr(&h_coo, &h_csr) != 0) {
         fprintf(stderr, "Error during COO to CSR conversion.\n");
         // Free memory and exit
         free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
         free(h_vec); free(h_res);
         free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);
         return -1;
    }

    printf("Finished Conversion from COO to CSR\n");

    // Free original COO data (host)
    free(h_coo.a_val); h_coo.a_val = NULL;
    free(h_coo.a_row); h_coo.a_row = NULL;
    free(h_coo.a_col); h_coo.a_col = NULL;

    // --- Calculate Optimal Launch Configuration ---
    int block_size, grid_size, shared_mem_size, kernel_variant = 0;

    calculate_advanced_launch_config(&h_csr, &block_size, &grid_size, &shared_mem_size, &kernel_variant);

    // --- Device Data Structures ---
    struct CSR d_csr; // Holds device pointers
    dtype *d_vec = NULL, *d_res = NULL;

    // --- Allocate Device Memory ---
    hipMalloc(&d_vec, m * sizeof(dtype));
    hipMalloc(&d_res, n * sizeof(dtype));
    hipMalloc(&d_csr.values, nnz * sizeof(dtype));
    hipMalloc(&d_csr.col_indices, nnz * sizeof(int));
    hipMalloc(&d_csr.row_pointers, (n + 1) * sizeof(int));
    
    d_csr.num_rows = n;
    d_csr.num_cols = m;
    d_csr.num_non_zeros = nnz;

    // Check for allocation errors
    hipError_t cuda_err = hipGetLastError();
    if (cuda_err != hipSuccess) {
        fprintf(stderr, "CUDA memory allocation failed: %s\n", hipGetErrorString(cuda_err));
        return -1;
    }

    // --- Copy Data to Device ---
    hipMemcpy(d_vec, h_vec, m * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_res, h_res, n * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.values, h_csr.values, nnz * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.col_indices, h_csr.col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.row_pointers, h_csr.row_pointers, (n + 1) * sizeof(int), hipMemcpyHostToDevice);

    // --- Timing Setup ---
    const int NUM_RUNS = 50;
    dtype total_time = 0.0;
    dtype times[NUM_RUNS];
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // --- Warmup Run ---
    vector_csr<<<grid_size, block_size>>>(
        d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n
    );
    hipDeviceSynchronize();
    
    printf("Launch config: Grid=%d, Block=%d, SharedMem=%d, Variant=%d\n", 
           grid_size, block_size, shared_mem_size, kernel_variant);

    hipError_t err;
    // --- Timed Runs ---
    for (int run = 0; run < NUM_RUNS; run++) {

        // Reset result vector before each run to ensure correctness
        hipMemset(d_res, 0, n * sizeof(dtype));

        switch (kernel_variant) {
            // case 1:
            //     hipEventRecord(start);
            //     vector_csr_shared_cache<<<grid_size,block_size,shared_mem_size>>>(
            //         d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n
            //     );
            //     hipEventRecord(end);
            //     hipEventSynchronize(end);
            //     break;
            case 2:
                hipEventRecord(start);
                vector_csr_double_buffer<<<grid_size,block_size>>>(
                    d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n
                );
                hipEventRecord(end);
                hipEventSynchronize(end);
                break;
            
            default:
                hipEventRecord(start);
                vector_csr<<<grid_size,block_size>>>(
                    d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n
                );
                hipEventRecord(end);
                hipEventSynchronize(end);
        }

        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
            return -1;
        }

        float millisec = 0.0;
        hipEventElapsedTime(&millisec, start, end);
        times[run] = millisec * 1e-3;
    }

    // --- Copy Result Back ---
    hipMemcpy(h_res, d_res, n * sizeof(dtype), hipMemcpyDeviceToHost);

    // --- Performance Calculation ---
    hipEventDestroy(start);
    hipEventDestroy(end);

    for (int i = 0; i < NUM_RUNS; i++) {
        total_time += times[i];
    }
    dtype avg_time = total_time / NUM_RUNS;

    // Calculate memory bandwidth more accurately for SpMV
    // For CSR SpMV, memory access pattern is:
    // 1. Read all row_pointers (accessed sequentially)
    // 2. Read all values and col_indices (accessed sequentially) 
    // 3. Read vector elements (potentially random access pattern)
    // 4. Write result vector (sequential)

    size_t bytes_read_vals = (size_t)nnz * sizeof(dtype);           // matrix values
    size_t bytes_read_cols = (size_t)nnz * sizeof(int);            // column indices  
    size_t bytes_read_row_ptr = (size_t)(n + 1) * sizeof(int);     // row pointers
    
    // For vector reads, each column index causes a vector element read
    // This gives a more realistic bandwidth estimate for custom kernels
    // that may not have sophisticated caching mechanisms
    size_t bytes_read_vec = (size_t)nnz * sizeof(dtype);           // vector reads (one per nnz)
    
    size_t bytes_written = (size_t)n * sizeof(dtype);              // result vector
    
    // Total memory traffic
    size_t total_bytes = bytes_read_vals + bytes_read_cols + 
                        bytes_read_row_ptr + bytes_read_vec + bytes_written;

    // Memory bandwidth calculation
    double bandwidth = total_bytes / (avg_time * 1.0e9);  // GB/s
    
    // Computational intensity
    double flops = 2.0 * nnz;  // Each non-zero: 1 multiply + 1 add
    double gflops = flops / (avg_time * 1.0e9);  // GFLOPS
    
    // Calculate arithmetic intensity for roofline analysis
    double arithmetic_intensity = flops / (double)total_bytes;  // FLOPS/Byte

    // --- Print Matrix Statistics ---
    print_matrix_stats(&h_csr);

    // --- Print Results with Additional Metrics ---
    printf("\n=== Vector CSR Performance Results ===\n");
    printf("Matrix: %s\n", argv[1]);
    printf("Dimensions: %d x %d, NNZ: %d\n", n, m, nnz);
    printf("Average time: %.6f seconds\n", avg_time);
    printf("Memory bandwidth: %.2f GB/s\n", bandwidth);
    printf("Compute performance: %.2f GFLOPS\n", gflops);
    printf("Arithmetic intensity: %.3f FLOPS/Byte\n", arithmetic_intensity);
    printf("Kernel variant used: %d\n", kernel_variant);
    printf("Memory breakdown:\n");
    printf("  Matrix values: %.2f MB\n", bytes_read_vals / (1024.0 * 1024.0));
    printf("  Column indices: %.2f MB\n", bytes_read_cols / (1024.0 * 1024.0));
    printf("  Row pointers: %.2f MB\n", bytes_read_row_ptr / (1024.0 * 1024.0));
    printf("  Vector reads: %.2f MB\n", bytes_read_vec / (1024.0 * 1024.0));
    printf("  Result writes: %.2f MB\n", bytes_written / (1024.0 * 1024.0));
    printf("  Total memory: %.2f MB\n", total_bytes / (1024.0 * 1024.0));

    // Also call the standard print function for consistency
    print_spmv_performance(
        "Vector CSR", 
        argv[1],
        n, 
        m, 
        nnz, 
        avg_time, 
        bandwidth, 
        gflops, 
        h_res,
        10  // Print up to 10 samples
    );

    // --- Cleanup ---
    hipFree(d_vec);
    hipFree(d_res);
    hipFree(d_csr.values);
    hipFree(d_csr.col_indices);
    hipFree(d_csr.row_pointers);

    free(h_vec);
    free(h_res);
    free(h_csr.values);
    free(h_csr.col_indices);
    free(h_csr.row_pointers);

    return 0;
}