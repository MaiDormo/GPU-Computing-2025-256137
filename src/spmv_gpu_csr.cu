#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <strings.h>
#include <wchar.h>
#include <time.h>
#include <sys/time.h>
#include <omp.h>
//#include <limits.h>

#include "../include/read_file_lib.h"
#include "../include/spmv_type.h"


#define WARP_SIZE 32
#define BLOCK_SIZE 1024

// --- Function Prototypes ---
int coo_to_csr(const struct COO *coo_data, struct CSR *csr_data);
__global__ void vector_csr(const dtype *csr_values, const int *csr_row_ptr, const int *csr_col_indices,
                                const dtype *vec, dtype *res, int n);


// --- COO to CSR Conversion ---
int coo_to_csr(const struct COO *coo_data, struct CSR *csr_data) {
    int n_rows = coo_data->num_rows;
    int nnz = coo_data->num_non_zeros;
    const int* a_row = coo_data->a_row;
    const int* a_col = coo_data->a_col;
    const dtype* a_val = coo_data->a_val;

    dtype* csr_values = csr_data->values;
    int* csr_col_indices = csr_data->col_indices;
    int* csr_row_ptr = csr_data->row_pointers;

    // Basic validation
    if (n_rows <= 0 || nnz < 0) return -1;
    if (nnz > 0 && (!a_row || !a_col || !a_val)) return -1;
    if (!csr_values || !csr_col_indices || !csr_row_ptr) return -1;

    // Initialize row pointers (assuming already zeroed by calloc)
    for (int i = 0; i < nnz; i++) {
        if(a_row[i] >= n_rows || a_row[i] < 0) {
             fprintf(stderr, "Error: Row index %d out of bounds (0-%d) at nnz index %d.\n", a_row[i], n_rows-1, i);
             return -1; // Invalid row index
        }
        csr_row_ptr[a_row[i] + 1]++;
    }

    for (int i = 0; i < n_rows; i++) {
        csr_row_ptr[i + 1] += csr_row_ptr[i];
    }

    int * temp_row_counts = (int *)calloc(n_rows, sizeof(int));
    if (!temp_row_counts) return -1;

    for (int i = 0; i < nnz; i++) {
        int row = a_row[i];
        int dest_indx = csr_row_ptr[row] + temp_row_counts[row];
        if (dest_indx >= nnz) {
             fprintf(stderr, "Error: Destination index %d out of bounds (%d) during CSR construction.\n", dest_indx, nnz);
             free(temp_row_counts);
             return -1; // Index out of bounds
        }
        csr_values[dest_indx] = a_val[i];
        csr_col_indices[dest_indx] = a_col[i];
        temp_row_counts[row]++;
    }
    free(temp_row_counts);

    // Sort within rows
    #pragma omp parallel for
    for (int i = 0; i < n_rows; i++) {
        int row_start = csr_row_ptr[i];
        int row_end = csr_row_ptr[i + 1];
        if (row_end <= row_start + 1) continue; // Skip empty or single-element rows

        for (int j = row_start + 1; j < row_end; j++) {
            int current_col = csr_col_indices[j];
            dtype current_val = csr_values[j];
            int k = j - 1;
            while (k >= row_start && csr_col_indices[k] > current_col) {
                csr_col_indices[k + 1] = csr_col_indices[k];
                csr_values[k + 1] = csr_values[k];
                k--;
            }
            csr_col_indices[k + 1] = current_col;
            csr_values[k + 1] = current_val;
        }
    }

    csr_data->num_rows = coo_data->num_rows;
    csr_data->num_cols = coo_data->num_cols;
    csr_data->num_non_zeros = coo_data->num_non_zeros;

    return 0;
}


// --- SpMV Kernel ---
__global__ void vector_csr(const dtype *csr_values, const int *csr_row_ptr, const int *csr_col_indices,
                                const dtype *vec, dtype *res, int n) { // n is num_rows

    const int tid = threadIdx.x;
    const int warps_per_block = BLOCK_SIZE / WARP_SIZE;
    const int block_start_row = blockIdx.x * warps_per_block;
    const int warp_id = tid / WARP_SIZE;
    const int lane_id = tid % WARP_SIZE;
    const int actual_row = block_start_row + warp_id;

    if (actual_row < n) {
        dtype thread_sum = 0.0;
        int row_start = csr_row_ptr[actual_row];
        int row_end = csr_row_ptr[actual_row + 1];

        for (int j = row_start + lane_id; j < row_end; j += WARP_SIZE) {
            int col = csr_col_indices[j];
            thread_sum += csr_values[j] * __ldg(&vec[col]);
        }

        #pragma unroll
        for (int delta = WARP_SIZE / 2; delta > 0; delta >>= 1) {
            thread_sum += __shfl_down_sync(0xFFFFFFFF, thread_sum, delta);
        }

        if (lane_id == 0) {
            res[actual_row] = thread_sum;
        }
    }
}


__global__ void vector_csr_unrolled(const dtype *csr_values, const int *csr_row_ptr, const int *csr_col_indices,
                                const dtype *vec, dtype *res, int n) { // n is num_rows

    const int tid = threadIdx.x;
    const int warps_per_block = BLOCK_SIZE / WARP_SIZE;
    const int block_start_row = blockIdx.x * warps_per_block;
    const int warp_id = tid / WARP_SIZE;
    const int lane_id = tid % WARP_SIZE;
    const int actual_row = block_start_row + warp_id;

    __shared__ volatile dtype vals[BLOCK_SIZE];

    if (actual_row < n) {
        dtype thread_sum = 0.0;
        int row_start = csr_row_ptr[actual_row];
        int row_end = csr_row_ptr[actual_row + 1];

        for (int j = row_start + lane_id; j < row_end; j += WARP_SIZE) {
            int col = csr_col_indices[j];
            thread_sum += csr_values[j] * __ldg(&vec[col]);
        }

        vals[tid] = thread_sum;
        __syncthreads();


        // Reduce partial sums loop unrolled
        if (lane_id < 16) vals[tid] += vals[tid + 16];
        if (lane_id < 8) vals[tid] += vals[tid + 8];
        if (lane_id < 4) vals[tid] += vals[tid + 4];
        if (lane_id < 2) vals[tid] += vals[tid + 2];
        if (lane_id < 1) vals[tid] += vals[tid + 1];
        __syncthreads();

        if (lane_id == 0) {
            res[actual_row] = vals[tid];
        }
    }
}


__global__ void adaptive_csr(const dtype *csr_values, const int *csr_row_ptr,
                            const int *csr_col_indices, const dtype *vec,
                            dtype *res, const int *row_blocks, int n) {
    //TODO implement
    /*
        General idea behind this implementation:
            -> warp per row:
            [PROS]: can be used as a lower bound of computation for the single row
            [CONS]: if a row contains a lot of NNZ values, the computation becomes slow
        
        To improve this concept, i asked myself these question: 
        - what if we can dispatch more warp per rows?
        - what is the minimal number of NNZ that a warp can handle
        - what could be the maximum number that a single warp should handle (possibly a multiple of the previous)?

        Implementing a good access pattern for vec is really hard, so for now i will use vector cache

    
    */
}

int adaptive_row_selection(const int *csr_row_ptr, int rows, int *row_blocks) {
    //TODO implement

    // inside this function i would like to compute how many rows are dispatched per block,
    // accounting for the fact that a warp is 32 threads and a block has 1024 threads, 
    // -> so at most a block can handle 8 rows!
}


// --- Main Function ---
int main(int argc, char ** argv) { 
    
    if (argc != 2) {
        printf("Usage: <./bin/spmv_*> <path/to/file.mtx>\n");
        return -1;
    }

    // --- Host Data Structures ---
    struct COO h_coo;
    struct CSR h_csr;
    dtype *h_vec = NULL;
    dtype *h_res = NULL;

    // --- Read Matrix ---
    read_from_file_and_init(argv[1], &h_coo);
    int n = h_coo.num_rows;
    int m = h_coo.num_cols;
    int nnz = h_coo.num_non_zeros;

    // --- Allocate Host Memory ---
    h_vec = (dtype*)malloc(m * sizeof(dtype));
    h_res = (dtype*)malloc(n * sizeof(dtype));
    h_csr.values = (dtype*)malloc(nnz * sizeof(dtype));
    h_csr.col_indices = (int*)malloc(nnz * sizeof(int));
    h_csr.row_pointers = (int*)calloc(n + 1, sizeof(int)); // Zero initialization is important
    int *h_block_rows = (int*)calloc(n, sizeof(int));


    if (!h_vec || !h_res || !h_csr.values || !h_csr.col_indices || !h_csr.row_pointers || !h_block_rows) {
        perror("Failed to allocate host memory");
        // Free any successfully allocated memory before exiting
        free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
        free(h_vec); free(h_res);
        free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);
        free(h_block_rows);
        return -1;
    }

    // --- Initialize Host Vectors ---
    for (int i = 0; i < m; i++) h_vec[i] = 1.0;
    memset(h_res, 0, n * sizeof(dtype));

    // --- Convert COO to CSR ---
    if (coo_to_csr(&h_coo, &h_csr) != 0) {
         fprintf(stderr, "Error during COO to CSR conversion.\n");
         // Free memory and exit
         free(h_coo.a_val); free(h_coo.a_row); free(h_coo.a_col);
         free(h_vec); free(h_res);
         free(h_csr.values); free(h_csr.col_indices); free(h_csr.row_pointers);
         free(h_block_rows);
         return -1;
    }

    // Free original COO data (host)
    free(h_coo.a_val); h_coo.a_val = NULL;
    free(h_coo.a_row); h_coo.a_row = NULL;
    free(h_coo.a_col); h_coo.a_col = NULL;

    int countRowBlocks = adaptive_row_selection(h_csr.row_pointers, n, h_block_rows);


    

    // --- Device Data Structures ---
    struct CSR d_csr; // Holds device pointers
    dtype *d_vec = NULL, *d_res = NULL;
    int * d_block_rows;

    // --- Allocate Device Memory ---
    hipMalloc(&d_vec, m * sizeof(dtype));
    hipMalloc(&d_res, n * sizeof(dtype));
    hipMalloc(&d_csr.values, nnz * sizeof(dtype));
    hipMalloc(&d_csr.col_indices, nnz * sizeof(int));
    hipMalloc(&d_csr.row_pointers, (n + 1) * sizeof(int));
    hipMalloc(&d_block_rows, countRowBlocks * sizeof(int));
    d_csr.num_rows = n;
    d_csr.num_cols = m;
    d_csr.num_non_zeros = nnz;

    // --- Copy Data to Device ---
    hipMemcpy(d_vec, h_vec, m * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_res, h_res, n * sizeof(dtype), hipMemcpyHostToDevice); // Copy initial zeros
    hipMemcpy(d_csr.values, h_csr.values, nnz * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.col_indices, h_csr.col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csr.row_pointers, h_csr.row_pointers, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_block_rows, h_block_rows, countRowBlocks * sizeof(int), hipMemcpyHostToDevice);

    // --- Kernel Launch Configuration ---
    const int warps_per_block = BLOCK_SIZE / WARP_SIZE;
    const int rows_per_block = warps_per_block / 2;
    const int block_num = (n + warps_per_block - 1) / warps_per_block;
    const size_t shared_mem = BLOCK_SIZE * sizeof(dtype); //!TODO configure shared_mem


    printf("Matrix dimensions: %d x %d with %d non-zeros\n", n, m, nnz);
    printf("Adaptive row blocks: %d\n", countRowBlocks);
    printf("Shared memory size: %zu bytes\n", shared_mem);

    // --- Timing Setup ---
    const int NUM_RUNS = 50;
    dtype total_time = 0.0;
    dtype times[NUM_RUNS];
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // --- Warmup Run ---
    // vector_csr_unrolled<<<block_num, thread_per_block, dynamic_shared_mem>>>(
        // d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n);
    vector_csr<<<block_num, BLOCK_SIZE, shared_mem>>>(
        d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n);
    
    hipDeviceSynchronize();

    // --- Timed Runs ---
    for (int run = 0; run < NUM_RUNS; run++) {
        hipEventRecord(start);

        // vector_csr_unrolled<<<block_num, thread_per_block, dynamic_shared_mem>>>(
            // d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n);
        
        vector_csr<<<block_num, BLOCK_SIZE, shared_mem>>>(
            d_csr.values, d_csr.row_pointers, d_csr.col_indices, d_vec, d_res, n);
            

        hipEventRecord(end);
        hipEventSynchronize(end);

        float millisec = 0.0;
        hipEventElapsedTime(&millisec, start, end);
        times[run] = millisec * 1e-3;
    }

    // --- Copy Result Back ---
    hipMemcpy(h_res, d_res, n * sizeof(dtype), hipMemcpyDeviceToHost);

    // --- Performance Calculation ---
    hipEventDestroy(start);
    hipEventDestroy(end); // Destroy events even if errors occurred after creation

    for (int i = 0; i < NUM_RUNS; i++) {
        total_time += times[i];
    }
    dtype avg_time = total_time / NUM_RUNS;

    size_t bytes_read = (size_t)nnz * (sizeof(dtype) + sizeof(int)) + // values and col indices
                        (size_t)(n + 1) * sizeof(int) +               // row pointers
                        (size_t)m * sizeof(dtype) +      // vector reads (worst case estimate)
                        (size_t)countRowBlocks * sizeof(int);
    size_t bytes_written = (size_t)n * sizeof(dtype);                 // result vector
    size_t total_bytes = bytes_read + bytes_written;

    double bandwidth = total_bytes / (avg_time * 1.0e9);  // GB/s
    double flops = 2.0 * nnz;
    double gflops = flops / (avg_time * 1.0e9);  // GFLOPS

    // --- Print Results ---
    printf("\nSpMV Performance CSR:\n");
    printf("Matrix size: %d x %d with %d non-zero elements\n", n, m, nnz);
    printf("Average execution time: %.3f ms\n", avg_time * 1.0e3);
    printf("Memory bandwidth (estimated): %.2f GB/s\n", bandwidth);
    printf("Computational performance: %.2f GFLOPS\n", gflops);

    printf("\nFirst few non-zero elements of result vector:\n");
    int count = 0;
    for (int i = 0; i < n && count < 10; i++) {
        if (h_res[i] != 0.0) {
            printf("%f ", h_res[i]);
            count++;
        }
    }
     if (count == 0) printf("Result vector is all zeros or first 10 elements are zero.");
    printf("\n");

    // --- Cleanup ---
    hipFree(d_vec);
    hipFree(d_res);
    hipFree(d_csr.values);
    hipFree(d_csr.col_indices);
    hipFree(d_csr.row_pointers);
    hipFree(d_block_rows);

    free(h_vec);
    free(h_res);
    free(h_csr.values);
    free(h_csr.col_indices);
    free(h_csr.row_pointers);
    free(h_block_rows);

    return 0;
}