#include "hip/hip_runtime.h"
#include "../include/spmv_kernels.h"

__global__ void spmv_simple(const dtype *csr_values, const int *csr_row_ptr, 
                          const int *csr_col_indices, const dtype *vec, 
                          dtype *res, int num_rows) {
    // Each thread processes one row
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < num_rows) {
        // Get the range of this row's elements
        int row_start = csr_row_ptr[row];
        int row_end = csr_row_ptr[row + 1];
        
        // Process each non-zero element in the row
        dtype sum = 0.0;
        for (int j = row_start; j < row_end; j++) {
            int col = csr_col_indices[j];
            sum += csr_values[j] * vec[col];
        }
        
        // Write the result
        res[row] = sum;
    }
}

__global__ void value_parallel_sequential_spmv(const dtype *csr_values, const int *csr_row_ptr, 
                              const int *csr_col_indices, const dtype *vec, 
                              dtype *res, int nnz, int num_rows) {
    // Each thread processes one non-zero value
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < nnz) {
        // Find which row this element belongs to (binary search)
        int row = 0;
        int left = 0;
        int right = num_rows - 1;
        
        while (left <= right) {
            int mid = (left + right) >> 1;
            if (csr_row_ptr[mid] <= idx) {
                row = mid;
                left = mid + 1;
            } else {
                right = mid - 1;
            }
        }
        
        // Get column and compute contribution
        int col = csr_col_indices[idx];
        dtype val = csr_values[idx] * vec[col];
        
        // Add contribution to result vector using atomic operation
        atomicAdd(&res[row], val);
    }
}

__global__ void value_parallel_blocked_spmv(const dtype *csr_values, const int *csr_row_ptr, 
    const int *csr_col_indices, const dtype *vec,
    dtype *res, int nnz, int num_rows, int stride) {
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate stride as total number of threads (distance between memory addresses accessed by sequential threads)
    
    int start_idx = tid * stride;
    
    // Simple strided access: each thread starts at its ID and jumps by stride
    for (int i = 0; i < stride; i++) {
        int idx = start_idx + i;
        if (idx >= nnz) break;
        // Find which row this element belongs to (binary search)
        int row = 0;
        int left = 0;
        int right = num_rows - 1;

        while (left <= right) {
            int mid = (left + right) >> 1;
            if (csr_row_ptr[mid] <= idx) {
                row = mid;
                left = mid + 1;
            } else {
                right = mid - 1;
            }
        }
        
        int col = csr_col_indices[idx];
        dtype val = csr_values[idx] * vec[col];
        atomicAdd(&res[row], val);
    }
}

//------------------- NOT PART OF THE FIRST DELIVERABLE --------------------------------

__global__ void vector_csr(const dtype *csr_values, const int *csr_row_ptr, const int *csr_col_indices,
    const dtype *vec, dtype *res, int n) { 
        const int tid = threadIdx.x;
        const int warps_per_block = BLOCK_SIZE / WARP_SIZE;
    const int block_start_row = blockIdx.x * warps_per_block;
    const int warp_id = tid / WARP_SIZE;
    const int lane_id = tid % WARP_SIZE;
    const int actual_row = block_start_row + warp_id;

    if (actual_row < n) {
        dtype thread_sum = 0.0;
        int row_start = csr_row_ptr[actual_row];
        int row_end = csr_row_ptr[actual_row + 1];

        for (int j = row_start + lane_id; j < row_end; j += WARP_SIZE) {
            int col = csr_col_indices[j];
            thread_sum += csr_values[j] * vec[col];
        }

        #pragma unroll
        for (int delta = WARP_SIZE >> 1; delta > 0; delta >>= 1) {
            thread_sum += __shfl_down_sync(0xFFFFFFFF, thread_sum, delta);
        }

        if (lane_id == 0) {
            res[actual_row] = thread_sum;
        }
    }
}

__global__ void vector_csr_unrolled(const dtype *csr_values, const int *csr_row_ptr, const int *csr_col_indices,
                                const dtype *vec, dtype *res, int n) {
    const int tid = threadIdx.x;
    const int warps_per_block = blockDim.x / WARP_SIZE;
    const int block_start_row = blockIdx.x * warps_per_block;
    const int warp_id = tid / WARP_SIZE;
    const int lane_id = tid % WARP_SIZE;
    const int actual_row = block_start_row + warp_id;

    __shared__ volatile dtype vals[BLOCK_SIZE];

    if (actual_row < n) {
        dtype thread_sum = 0.0;
        int row_start = csr_row_ptr[actual_row];
        int row_end = csr_row_ptr[actual_row + 1];

        for (int j = row_start + lane_id; j < row_end; j += WARP_SIZE) {
            int col = csr_col_indices[j];
            thread_sum += csr_values[j] * __ldg(&vec[col]);
        }

        vals[tid] = thread_sum;
        __syncwarp(); // Synchronize only within the warp before reduction

        // Reduce partial sums loop unrolled
        // No synchronization needed between these steps within a warp
        if (lane_id < 16) vals[tid] += vals[tid + 16];
        if (lane_id < 8) vals[tid] += vals[tid + 8];
        if (lane_id < 4) vals[tid] += vals[tid + 4];
        if (lane_id < 2) vals[tid] += vals[tid + 2];
        if (lane_id < 1) vals[tid] += vals[tid + 1];
        
        // Only need this final syncwarp to ensure reduction is complete before writing result
        __syncwarp();

        if (lane_id == 0) {
            res[actual_row] = vals[tid];
        }
    }
}

__global__ void adaptive_csr(const dtype *csr_values, const int *csr_row_ptr,
                            const int *csr_col_indices, const dtype *vec,
                            dtype *res, const int *row_blocks, int n) {
   __shared__ volatile dtype SHARED_MEM[BLOCK_SIZE];

    const int tid = threadIdx.x;
    const int WARP_NUM = tid / WARP_SIZE;
    const int WARP_ID = tid % WARP_SIZE;

    int block_row_start = row_blocks[blockIdx.x];
    int block_row_end = row_blocks[blockIdx.x + 1];
    int num_rows = block_row_end - block_row_start;
    if (num_rows > 1) {
        //that means we have a warp per row
        int warp_row_index = block_row_start + WARP_NUM;
        if (warp_row_index < block_row_end) {
            int warp_row_start = csr_row_ptr[warp_row_index];
            int warp_row_end = csr_row_ptr[warp_row_index + 1];
            dtype thread_sum = 0.0;
            
            for (int i = warp_row_start + WARP_ID; i < warp_row_end; i += WARP_SIZE) {
                thread_sum += csr_values[i] * __ldg(&vec[csr_col_indices[i]]);
            }

            //warp reduction
            #pragma unroll
            for (int delta = WARP_SIZE / 2; delta > 0; delta >>= 1) {
                thread_sum += __shfl_down_sync(0xFFFFFFFF, thread_sum, delta);
            }

            if (WARP_ID == 0) {
                res[warp_row_index] = thread_sum;
            }
        }
        
    } else {
        //that means we have a block per row
        int row_idx = block_row_start;
        int row_start = csr_row_ptr[block_row_start];
        int row_end = csr_row_ptr[block_row_end];
        dtype thread_sum = 0.0;
        for (int i = row_start + tid; i < row_end; i += blockDim.x) {
            thread_sum += csr_values[i] * __ldg(&vec[csr_col_indices[i]]);
        }

        //warp reduction
        #pragma unroll
        for (int delta = WARP_SIZE / 2; delta > 0; delta >>= 1) {
            thread_sum += __shfl_down_sync(0xFFFFFFFF, thread_sum, delta);
        }

        //one thread of each warp saves on shared memory
        if (WARP_ID == 0) {
            SHARED_MEM[WARP_NUM] = thread_sum;
        }

        if (tid < WARP_SIZE) {
            #pragma unroll
            for (int delta = WARP_SIZE / 2; delta > 0; delta >>= 1) {
                thread_sum += __shfl_down_sync(0xFFFFFFFF, thread_sum, delta);
            }

            if (tid == 0)
                res[row_idx] = thread_sum;
        }
    }
}


